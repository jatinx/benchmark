#include "hip/hip_runtime.h"
#define ENABLEGPU
#include <benchmark/benchmark.h>

__global__ void add(int* a) { *a += 3; }

static void BM_CudaMalloc(benchmark::State& state) {
  for (auto _ : state) {
    BENCHMARK_GPU_DECLARE();
    BENCHMARK_GPU_PRE_KERNEL();
    int *d_a;
    hipMalloc(&d_a, sizeof(int));
    hipFree(d_a);
    BENCHMARK_GPU_POST_KERNEL();
    BENCHMARK_GPU_SET_TIME();
    BENCHMARK_GPU_CLEANUP();
  }
}
// Register the function as a benchmark
BENCHMARK(BM_CudaMalloc)->UseManualTime();

// Define another benchmark
static void BM_LaunchKernel(benchmark::State& state) {
  for (auto _ : state) {
    BENCHMARK_GPU_DECLARE();
    BENCHMARK_GPU_PRE_KERNEL();
    int *d_a;
    hipMalloc(&d_a, sizeof(int));
    add<<<1,1>>>(d_a);
    hipFree(d_a);
    BENCHMARK_GPU_POST_KERNEL();
    BENCHMARK_GPU_SET_TIME();
    BENCHMARK_GPU_CLEANUP();
  }
}
BENCHMARK(BM_LaunchKernel)->UseManualTime();

BENCHMARK_MAIN();
